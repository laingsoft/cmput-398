#include "hip/hip_runtime.h"
#include <wb.h>
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>

#define NUM_BINS 4096

#define CUDA_CHECK(ans)                                                   \
    { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
	bool abort = true) {
	if (code != hipSuccess) {
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code),
			file, line);
		if (abort)
			exit(code);
	}
}


__global__ void histogramKernel(unsigned int *input, unsigned int *bins, int inputLength){
	unsigned int tid = blockIdx.x*blockDim.x + threadIdx.x;

	__shared__ unsigned int histo_s[NUM_BINS];

	for (unsigned int binIdx = threadIdx.x; binIdx < NUM_BINS; binIdx += blockDim.x){
		histo_s[binIdx] = 0;
	}

	__syncthreads();

	for (unsigned int i = tid; i < inputLength; i += blockDim.x*gridDim.x){
		atomicAdd(&(histo_s[input[i]]),1);
	}
	__syncthreads();
	for (unsigned int binIdx = threadIdx.x; binIdx < NUM_BINS; binIdx += blockDim.x){
		atomicAdd(&(bins[binIdx]), histo_s[binIdx]);
	}
	


}

__global__ void histClean(unsigned int *bins){
	int id = blockIdx.x*blockDim.x + threadIdx.x;

	if (id <= NUM_BINS){
		if (bins[id] > 127) bins[id] = 127;
	}

}





int main(int argc, char *argv[]) {
	wbArg_t args;
	int inputLength;
	unsigned int *hostInput;
	unsigned int *hostBins;
	unsigned int *deviceInput;
	unsigned int *deviceBins;

	args = wbArg_read(argc, argv);

	wbTime_start(Generic, "Importing data and creating memory on host");
	hostInput = (unsigned int *)wbImport(wbArg_getInputFile(args, 0),
		&inputLength, "Integer");
	hostBins = (unsigned int *)malloc(NUM_BINS * sizeof(unsigned int));
	wbTime_stop(Generic, "Importing data and creating memory on host");

	wbLog(TRACE, "The input length is ", inputLength);
	wbLog(TRACE, "The number of bins is ", NUM_BINS);

	wbTime_start(GPU, "Allocating GPU memory.");
	// TODO: Allocate GPU memory here
	hipMalloc(&deviceInput, inputLength*sizeof(unsigned int));
	hipMalloc(&deviceBins, NUM_BINS*sizeof(unsigned int));

	CUDA_CHECK(hipDeviceSynchronize());
	wbTime_stop(GPU, "Allocating GPU memory.");

	wbTime_start(GPU, "Copying input memory to the GPU.");
	// TODO: Copy memory to the GPU here
	hipMemcpy(deviceInput, hostInput, inputLength*sizeof(unsigned int),hipMemcpyHostToDevice);
	//hipMemcpy(deviceBins, hostBins, NUM_BINS*sizeof(unsigned int), hipMemcpyHostToDevice);

	CUDA_CHECK(hipDeviceSynchronize());
	wbTime_stop(GPU, "Copying input memory to the GPU.");

	// Launch kernel
	// ----------------------------------------------------------
	int blocksize = 16;
	dim3 dimGrid(ceil(inputLength/16)+1, 1, 1);
	dim3 dimBlock(blocksize, 1, 1);

	wbLog(TRACE, "Launching kernel");
	wbTime_start(Compute, "Performing CUDA computation");

	// TODO: Perform kernel computation here

	histogramKernel << <dimGrid, dimBlock >> >(deviceInput, deviceBins, inputLength);
	histClean << <dimGrid, dimBlock >> > (deviceBins);

	// You should call the following lines after you call the kernel.
	CUDA_CHECK(hipGetLastError());
	CUDA_CHECK(hipDeviceSynchronize());

	wbTime_stop(Compute, "Performing CUDA computation");

	wbTime_start(Copy, "Copying output memory to the CPU");
	// TODO: Copy the GPU memory back to the CPU here
	hipMemcpy(hostBins, deviceBins, NUM_BINS*sizeof(unsigned int), hipMemcpyDeviceToHost);
	CUDA_CHECK(hipDeviceSynchronize());
	wbTime_stop(Copy, "Copying output memory to the CPU");


	wbTime_start(GPU, "Freeing GPU Memory");
	// TODO: Free the GPU memory here
	hipFree(deviceBins);
	hipFree(deviceInput);


	wbTime_stop(GPU, "Freeing GPU Memory");

	// Verify correctness
	// -----------------------------------------------------
	wbSolution(args, hostBins, NUM_BINS);

	free(hostBins);
	free(hostInput);

#if LAB_DEBUG
	system("pause");
#endif

	return 0;
}

#include "hip/hip_runtime.h"
// Given a list (lst) of length n
// Output its sum = lst[0] + lst[1] + ... + lst[n-1];

#include "hip/hip_runtime.h"
#include ""

#include <wb.h>

#define BLOCK_SIZE 512 // TODO: You can change this

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      return -1;                                                          \
		    }                                                                     \
      } while (0)

__global__ void simple_total(float *input, float *output, int len) {
	__shared__ float partialSum[2 * BLOCK_SIZE];
	unsigned int t = threadIdx.x;
	unsigned int start = 2 * blockIdx.x*BLOCK_SIZE;
	partialSum[t] = input[start + t];
	partialSum[BLOCK_SIZE + t] = input[start + BLOCK_SIZE + t];

	for (unsigned int stride = 1; stride <= blockDim.x; stride *= 2)
	{
		__syncthreads();
		if (t % stride == 0)
			partialSum[2 * t] += partialSum[2 * t + stride];
	}

	if (t == 0)
		output[blockIdx.x] = partialSum[0];
}

int main(int argc, char **argv) {
	wbArg_t args;
	float *hostInput;  // The input 1D list
	float *hostOutput; // The output list
	float *deviceInput;
	float *deviceOutput;
	int numInputElements;  // number of elements in the input list
	int numOutputElements; // number of elements in the output list

	args = wbArg_read(argc, argv);

	wbTime_start(Generic, "Importing data and creating memory on host");
	hostInput =
		(float *)wbImport(wbArg_getInputFile(args, 0), &numInputElements);

	numOutputElements = numInputElements / (BLOCK_SIZE << 1);
	if (numInputElements % (BLOCK_SIZE << 1)) {
		numOutputElements++;
	}
	hostOutput = (float *)malloc(numOutputElements * sizeof(float));

	wbTime_stop(Generic, "Importing data and creating memory on host");

	wbLog(TRACE, "The number of input elements in the input is ",
		numInputElements);
	wbLog(TRACE, "The number of output elements in the input is ",
		numOutputElements);

	wbTime_start(GPU, "Allocating GPU memory.");
	// TODO: Allocate GPU memory here
	hipMalloc(&deviceInput, sizeof(float)*numInputElements);
	hipMalloc(&deviceOutput, sizeof(float)*numOutputElements);

	wbTime_stop(GPU, "Allocating GPU memory.");

	wbTime_start(GPU, "Copying input memory to the GPU.");
	// TODO: Copy memory to the GPU here
	hipMemcpy(deviceInput, hostInput, sizeof(float)*numInputElements, hipMemcpyHostToDevice);

	wbTime_stop(GPU, "Copying input memory to the GPU.");
	// TODO: Initialize the grid and block dimensions here
	int blocksize = BLOCK_SIZE;
	dim3 dimGrid((numInputElements+blocksize-1)/blocksize, 1, 1);
	dim3 dimBlock(blocksize,1,1);


	wbTime_start(Compute, "Performing CUDA computation");
	// TODO: Launch the GPU Kernel here
	simple_total << <dimGrid, dimBlock >> >(deviceInput, deviceOutput, numInputElements);
	hipDeviceSynchronize();
	wbTime_stop(Compute, "Performing CUDA computation");

	wbTime_start(Copy, "Copying output memory to the CPU");
	// TODO: Copy the GPU memory back to the CPU here
	hipMemcpy(hostOutput, deviceOutput, sizeof(float)*numOutputElements, hipMemcpyDeviceToHost);

	wbTime_stop(Copy, "Copying output memory to the CPU");

	// TODO:
	/********************************************************************
	* Reduce output vector on the host
	* NOTE: One could also perform the reduction of the output vector
	* recursively and support any size input. For simplicity, we do not
	* require that for this lab.
	********************************************************************/
	for (unsigned int i = 0; i < numOutputElements-1; i++) hostOutput[0] += hostOutput[i];

	wbTime_start(GPU, "Freeing GPU Memory");
	// TODO: Free the GPU memory here
	hipFree(deviceInput);
	hipFree(deviceOutput);
	wbTime_stop(GPU, "Freeing GPU Memory");

	wbSolution(args, hostOutput, 1);

	free(hostInput);
	free(hostOutput);

#if LAB_DEBUG
	system("pause");
#endif

	return 0;
}
